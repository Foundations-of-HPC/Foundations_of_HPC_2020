#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void vector_add(int *a, int *b, int *c)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = /* FIXME */
	c[index] = a[index] + b[index];
}

/* experiment with N */
/* how large can it be? */
#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main()
{
  int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	int size = N * sizeof( int );

	/* allocate space for device copies of a, b, c */
	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_b, size );
	hipMalloc( (void **) &d_c, size );

	/* allocate space for host copies of a, b, c and setup input values */

	a = (int *)malloc( size );
	b = (int *)malloc( size );
	c = (int *)malloc( size );

	for( int i = 0; i < N; i++ )
	{
		a[i] = b[i] = i;
		c[i] = 0;
	}

	/* copy inputs to device */
	/* fix the parameters needed to copy data to the device */
	hipMemcpy( /* FIXME */ );
	hipMemcpy( /* FIXME */ );

	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	add<<< /* FIXME */, /* FIXME */ >>>( d_a, d_b, d_c );

	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	hipMemcpy( /* FIXME */ );


	printf( "c[0] = %d\n",0,c[0] );
	printf( "c[%d] = %d\n",N-1, c[N-1] );

	/* clean up */

	free(a);
	free(b);
	free(c);
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	
	return 0;
} /* end main */
